#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define THREADS 30
#define BLOCKS 200
#define SIZE 6000

__device__ float d_A[SIZE][SIZE];
__device__ float d_B[SIZE][SIZE];
__device__ float d_C[SIZE][SIZE];
__device__ float d_D[SIZE][SIZE];
__device__ float d_V[SIZE];
__device__ float d_VET[SIZE];
__device__ float ESCALAR = 1.25;

__global__ void load()
{
    for(int i = 0; i < SIZE; i++)
    {
        for(int j = 0; j < SIZE; j++) 
        {
            d_A[i][j] = i + j;
            d_B[i][j] = i + j;
            d_C[i][j] = 0;
            d_D[i][j] = 0;
        }

        d_V[i] = i;
        d_VET[i] = 0;
    }
}

__global__ void sumA_B()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i < SIZE && j < SIZE)
        d_C[i][j] = d_A[i][j] + d_B[i][j];
}

__global__ void mulA_B()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i < SIZE && j < SIZE)
    {
        for(int k = 0; k < SIZE; k++) 
            d_D[i][j] += d_A[i][k] * d_B[k][j];
    }
}

__global__ void mulA_ESCALAR()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i < SIZE && j < SIZE)
        d_A[i][j] *= ESCALAR;
}

__global__ void mulB_V()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < SIZE)
    {
        for(int j = 0; j < SIZE; j++)
            d_VET[i] += d_B[i][j] * d_V[j];
    }
}

int main()
{  
    clock_t begin, end;

    hipSetDevice(0);  
    load<<<1, 1>>>();
    hipDeviceSynchronize();
    
    printf("Somar A e B e armazenar em C.\n");
    begin = clock();
    sumA_B<<<dim3(BLOCKS, BLOCKS), dim3(THREADS, THREADS)>>>();
    hipDeviceSynchronize();
    end = clock();
    printf("Feito em %.3f segundos.\n", double(end - begin) / CLOCKS_PER_SEC);

    printf("Multiplicar A e B e armazenar em D.\n");
    begin = clock();
    mulA_B<<<dim3(BLOCKS, BLOCKS), dim3(THREADS, THREADS)>>>();
    hipDeviceSynchronize();
    end = clock();
    printf("Feito em %.3f segundos.\n", double(end - begin) / CLOCKS_PER_SEC);

    printf("Multiplicar A e ESCALAR e armazenar em A.\n");
    begin = clock();
    mulA_ESCALAR<<<dim3(BLOCKS, BLOCKS), dim3(THREADS, THREADS)>>>();
    hipDeviceSynchronize();
    end = clock();
    printf("Feito em %.3f segundos.\n", double(end - begin) / CLOCKS_PER_SEC);
    
    printf("Multiplicar B e V e armazenar em VET.\n");
    begin = clock();
    mulB_V<<<BLOCKS, THREADS>>>();    
    hipDeviceSynchronize();
    end = clock();
    printf("Feito em %.3f segundos.\n", double(end - begin) / CLOCKS_PER_SEC);
}